
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double) (tp.tv_sec + tp.tv_usec*1e-6);
}
void Write(double* buffer, int np, char* output){
   FILE *f;
   f=fopen(output,"w");
    for(int i=0;i<np;++i){
       fprintf(f,"%f \n",buffer[i]);
    }
    fclose(f);
}
void force_repulsion(int np, const double *pos, double L, double krepulsion, 
    double *forces)
{
    int i, j;
    double posi[4];
    double rvec[4];
    double s2, s, f;

    // initialize forces to zero
    for (i=0; i<3*np; i++)
        forces[i] = 0.;

    // loop over all pairs
    for (i=0; i<np; i++)
    {
        posi[0] = pos[3*i  ];
        posi[1] = pos[3*i+1];
        posi[2] = pos[3*i+2];

        for (j=i+1; j<np; j++)
        {
            // compute minimum image difference
            rvec[0] = remainder(posi[0] - pos[3*j  ], L);
            rvec[1] = remainder(posi[1] - pos[3*j+1], L);
            rvec[2] = remainder(posi[2] - pos[3*j+2], L);

            s2 = rvec[0]*rvec[0] + rvec[1]*rvec[1] + rvec[2]*rvec[2];

            if (s2 < 4)
            {
                s = sqrt(s2);
                rvec[0] /= s;
                rvec[1] /= s;
                rvec[2] /= s;
                f = krepulsion*(2.-s);

                forces[3*i  ] +=  f*rvec[0];
                forces[3*i+1] +=  f*rvec[1];
                forces[3*i+2] +=  f*rvec[2];
                forces[3*j  ] += -f*rvec[0];
                forces[3*j+1] += -f*rvec[1];
                forces[3*j+2] += -f*rvec[2];
            }
        }
    }
}

__global__ void gpu_find_repulsion(int np, double*pos, double L, double krepulsion, double* forces){
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     if(i<np){
     	int j;
     	double posi[4];
     	double rvec[4];
     	double s2, s, f;
     	posi[0] = pos[3*i  ];
     	posi[1] = pos[3*i+1];
     	posi[2] = pos[3*i+2];
	 	for (j=i+1; j<np; j++){
        // compute minimum image difference
         	rvec[0] = remainder(posi[0] - pos[3*j  ], L);
         	rvec[1] = remainder(posi[1] - pos[3*j+1], L);
         	rvec[2] = remainder(posi[2] - pos[3*j+2], L);
         	s2 = rvec[0]*rvec[0] + rvec[1]*rvec[1] + rvec[2]*rvec[2];
         	if (s2 < 4){
             	s = sqrt(s2);
             	rvec[0] /= s;
             	rvec[1] /= s;
             	rvec[2] /= s;
             	f = krepulsion*(2.-s);
             	forces[3*i  ] +=  f*rvec[0];
             	forces[3*i+1] +=  f*rvec[1];
                forces[3*i+2] +=  f*rvec[2];
             	forces[3*j  ] += -f*rvec[0];
             	forces[3*j+1] += -f*rvec[1];
             	forces[3*j+2] += -f*rvec[2];
            }
        }
     }
}


int main(int argc, char *argv[])
{
    int i;
    int np = 100;             // default number of particles
    double phi = 0.3;         // volume fraction
    double krepulsion = 125.; // force constant
    double *pos;
    double *forces;
    double time0, time1;

    if (argc > 1)
        np = atoi(argv[1]);

    // compute simulation box width
    double L = pow(4./3.*3.1415926536*np/phi, 1./3.);

    // generate random particle positions inside simulation box
    forces = (double *) malloc(3*np*sizeof(double));
    pos    = (double *) malloc(3*np*sizeof(double));
    for (i=0; i<3*np; i++)
        pos[i] = rand()/(double)RAND_MAX*L;
    time0 = get_walltime();
    force_repulsion(np, pos, L, krepulsion, forces);
    time1 = get_walltime();
    //print performance and write to file
    printf("number of particles: %d\n", np);
    printf("elapsed time of cpu program: %f seconds\n", time1-time0);
    Write(forces,3*np,"cpu_output"); 
    //reinitialization of forces
    for(int i=0;i<np*3;++i) forces[i]=0.;
    //gpu program
    double *gpu_pos;
    double *gpu_forces;
    int bytes=3*np*sizeof(double);
    hipEvent_t start, stop;
    float time;
    hipMalloc((void**)&gpu_pos,bytes);
    hipMalloc((void**)&gpu_forces,bytes);
    hipMemcpy(gpu_pos, pos, bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_forces, forces, bytes, hipMemcpyHostToDevice);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    gpu_find_repulsion<<<(3*np+1023)/1024,1024>>>(np, gpu_pos, L, krepulsion, gpu_forces);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipMemcpy(forces, gpu_forces, bytes, hipMemcpyDeviceToHost);
    printf("number of particles: %d\n", np);
    printf("elapsed time of gpu program: %f seconds\n", time/1000);
    Write(forces,3*np,"gpu_output");
    printf("speed up of gpu is %f \n",(time1-time0)/(time/1000));
    hipFree(gpu_pos);
    hipFree(gpu_forces);
    free(forces);
    free(pos);

    return 0;
}
